#include "hip/hip_runtime.h"
extern "C"
{
#include <lualib.h>
#include <lauxlib.h>
#include <lua.h>
}

//#include "THCUNN.h"
#include "common.h"
#include "stdio.h"
#include <THC/THC.h> 
#include <THC/THCApply.cuh>

#define ELEMENT(array, i, j, k)  array[((i) * ((rowsCount) * (columnsCount))) + ((j) * (columnsCount)) + (k)]

extern "C" {
    
__global__ void _calcNeighborhoodDifference(const float* input, int layersCount, int rowsCount, int columnsCount, float* output1, float* output2) {

	// get the current layer and element indices
	int currentLayerIndex = blockIdx.x;
	int currentElementIndex = threadIdx.x;

	//printf("element(%d) = %f, element(%d) = %f\n", currentElementIndex, input[currentElementIndex], 12 + currentElementIndex, input[12 + currentElementIndex]);

	// get the correct row and column indices
	int halfLayersCount = layersCount / 2;
	int currentRow = currentElementIndex / columnsCount;
	int currentColumn = currentElementIndex % columnsCount;

	// get the elements that are in focus now
	float focusElementInA = ELEMENT(input, currentLayerIndex, currentRow, currentColumn);
	float focusElementInB = ELEMENT(input, currentLayerIndex + halfLayersCount, currentRow, currentColumn);

	// calculate the neighborhood differences
	for(int xIndex = -2 ; xIndex <= 2; xIndex++) {
		for(int yIndex = -2 ; yIndex <= 2; yIndex++) {
			// A - B calculation
			int currentXIndex = currentRow + xIndex;
			int currentYIndex = currentColumn + yIndex;
			float elementInA = 0, elementInB = 0;

			int outputLayerIndex = (currentLayerIndex * 25)+ (xIndex + 2) * 5 + (yIndex + 2);

			if(currentXIndex >= 0 && currentXIndex < rowsCount && currentYIndex >= 0 && currentYIndex < columnsCount)
			{
				elementInA = ELEMENT(input, currentLayerIndex, currentXIndex, currentYIndex);
				elementInB = ELEMENT(input, currentLayerIndex + halfLayersCount, currentXIndex, currentYIndex);
				//printf("element(%d, %d, %d) = %f, element(%d, %d, %d) = %f\n", currentLayerIndex, currentXIndex, currentYIndex, elementInA, currentLayerIndex + halfLayersCount, currentXIndex, currentYIndex, elementInB);
				//printf("\n", );
			}

			float neighborhoodDiffAtoB = focusElementInA - elementInB;
			float neighborhoodDiffBtoA = focusElementInB - elementInA;

			// place the element in correct position
			ELEMENT(output1, outputLayerIndex, currentRow, currentColumn) = neighborhoodDiffAtoB;
			ELEMENT(output2, outputLayerIndex, currentRow, currentColumn) = neighborhoodDiffBtoA;
		}
	}
}

void CIN_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *output1, THCudaTensor *output2) {
    THCUNN_assertSameGPU(state, 3, input, output1, output2);

	int inputLayers = input->size[0];
	int inputWidth = input->size[1];
	int inputHeight = input->size[2];

	float* inputContents = THCudaTensor_data(state, input);

	//resize output
	//for each layer, 25 neighbors (5 x 5) of each pixel
	THCudaTensor_resize3d(state, output1, (inputLayers / 2) * 25, inputWidth, inputHeight);
	THCudaTensor_resize3d(state, output2, (inputLayers / 2) * 25, inputWidth, inputHeight);
	float* outputPtr1 = THCudaTensor_data(state, output1);
	float* outputPtr2 = THCudaTensor_data(state, output2);

	_calcNeighborhoodDifference<<<inputLayers/2, inputWidth * inputHeight>>>(inputContents, inputLayers, inputWidth, inputHeight, outputPtr1, outputPtr2);

}


__global__ void _calcGradInput(const float* input, int layersCount, int rowsCount, int columnsCount, float* gradOutput1, float* gradOutput2, float* gradInput)
{

	// get the current layer and element indices
	int currentLayerIndex = blockIdx.x;
	int currentElementIndex = threadIdx.x;

	//printf("element(%d) = %f, element(%d) = %f\n", currentElementIndex, input[currentElementIndex], 12 + currentElementIndex, input[12 + currentElementIndex]);

	// get the correct row and column indices
	int halfLayersCount = layersCount / 2;
	int currentRow = currentElementIndex / columnsCount;
	int currentColumn = currentElementIndex % columnsCount;

	// get the elements that are in focus now
	float gradForElementInA = 0;
	float gradForElementInB = 0;

	// calculate the neighborhood differences
	for(int xIndex = -2 ; xIndex <= 2; xIndex++) {
		for(int yIndex = -2 ; yIndex <= 2; yIndex++) {
			int outputLayerIndex = (currentLayerIndex * 25)+ (xIndex + 2) * 5 + (yIndex + 2);

			// add positive gradients (independent of currentXIndex, currentYIndex), only dependent on outputLayerIndex
			gradForElementInA += ELEMENT(gradOutput1, outputLayerIndex, currentRow, currentColumn);
			gradForElementInB += ELEMENT(gradOutput2, outputLayerIndex, currentRow, currentColumn);

			// subtract gradients from gradOutput2 (or gradOutput1)
			// to find the correct gradOutput element of current layer, refer below

			//0  0  0  0  0
			//0  0  0  0  0
			//0  0  x  0  0
			//10 9  8  7  6
			//5  4  3  2  1

			int currentNegGradXIndex = currentRow - xIndex;
			int currentNegGradYIndex = currentColumn - yIndex;

			if(currentNegGradXIndex >= 0 && currentNegGradXIndex < rowsCount && currentNegGradYIndex >= 0 && currentNegGradYIndex < columnsCount)
			{
				gradForElementInA -= ELEMENT(gradOutput2, outputLayerIndex, currentNegGradXIndex, currentNegGradYIndex);
				gradForElementInB -= ELEMENT(gradOutput1, outputLayerIndex, currentNegGradXIndex, currentNegGradYIndex);
			}
		}
	}

	ELEMENT(gradInput, currentLayerIndex, currentRow, currentColumn) = gradForElementInA;
	ELEMENT(gradInput, currentLayerIndex + halfLayersCount, currentRow, currentColumn) = gradForElementInB;
}


void CIN_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *gradOutput1, THCudaTensor *gradOutput2, THCudaTensor *gradInput) {

	//determine gradInput sizes
	int layersCount = input->size[0];
	int rowsCount = input->size[1];
	int columnsCount = input->size[2];

	//resize gradInputs
	THCudaTensor_resize3d(state, gradInput, layersCount, rowsCount, columnsCount);

	//get elementary datatype pointers
	float* inputContents = THCudaTensor_data(state, input);
	float* gradOutputPtr1 = THCudaTensor_data(state, gradOutput1);
	float* gradOutputPtr2 = THCudaTensor_data(state, gradOutput2);
	float* gradInputPtr = THCudaTensor_data(state, gradInput);

	//calculate gradient of final output with respect to each input element
	_calcGradInput<<<layersCount/2, rowsCount * columnsCount>>>(inputContents, layersCount, rowsCount, columnsCount, gradOutputPtr1, gradOutputPtr2, gradInputPtr);

}

}
