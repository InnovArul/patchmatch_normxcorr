#include "hip/hip_runtime.h"
extern "C"
{
#include <lualib.h>
#include <lauxlib.h>
#include <lua.h>
}

//#include "THCUNN.h"
#include "common.h"
#include "stdio.h"
#include <THC/THC.h> 
#include <THC/THCApply.cuh>

#define ELEMENT(array, i, j, k, limitHeight, limitWidth)  (array[((i) * ((limitHeight) * (limitWidth))) + ((j) * (limitWidth)) + (k)])
#define CHECKIN  printf("At line number : %d : %s \n", __LINE__, __FILE__)

// Define this to turn on error checking
#define CUDA_ERROR_CHECK
//#define DEBUG_LEVEL 1
#define MALLOC_LIMIT  2047*1024*1024

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )
#define MAX_THREADS 144

extern "C" {

bool NormCrossMapCorrelationSmallerSearch_IsMallocSet = false;


/**
 * API to call Cuda APIs safely
 * @param err
 * @param file
 * @param line
 */
inline void __cudaSafeCall(hipError_t err, const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
	if (hipSuccess != err) {
		fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line,
				hipGetErrorString(err));
		exit(-1);
	}
#endif

	return;
}

/**
 * API to check the last returned cuda error
 * @param file
 * @param line
 */
inline void __cudaCheckError(const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n", file, line,
				hipGetErrorString(err));
		exit(-1);
	}

	// More careful checking. However, this will affect performance.
	// Comment away if needed.
	err = hipDeviceSynchronize();
	if (hipSuccess != err) {
		fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
				file, line, hipGetErrorString(err));
		exit(-1);
	}
#endif

	return;
}

/**
 * API to report the memory usage of the GPU
 */
static void reportMemStatus() {

	// show memory usage of GPU
	size_t free_byte;
	size_t total_byte;
	size_t malloc_byte;

	hipError_t cuda_status = hipMemGetInfo(&free_byte, &total_byte);

	if (hipSuccess != cuda_status) {
		printf("Error: hipMemGetInfo fails, %s \n",
				hipGetErrorString(cuda_status));
		return;
	}

	cuda_status = hipDeviceGetLimit(&malloc_byte, hipLimitMallocHeapSize);
	if (hipSuccess != cuda_status) {
			printf("Error: hipDeviceGetLimit fails, %s \n",
					hipGetErrorString(cuda_status));
			return;
	}

	double free_db = (double) free_byte;
	double total_db = (double) total_byte;
	double used_db = total_db - free_db;
	printf("GPU memory usage: used = %f, free = %f MB, total = %f MB, malloc limit = %f MB\n",
			used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0,
			total_db / 1024.0 / 1024.0, malloc_byte / 1024.0 / 1024.0);

}

/**
 * API to set the malloc limit of GPU
 */
static void setMallocLimit() {

	// hipDeviceSetLimit can be called only once to set the malloc limit
	// this if loop is to prevent multiple calls of hipDeviceSetLimit
	if(!NormCrossMapCorrelationSmallerSearch_IsMallocSet)
	{
		hipError_t	cuda_status = hipDeviceSetLimit(hipLimitMallocHeapSize, MALLOC_LIMIT);
		if (hipSuccess != cuda_status) {
				printf("Error: hipDeviceSetLimit fails, %s \n",
						hipGetErrorString(cuda_status));
				return;
		}

		NormCrossMapCorrelationSmallerSearch_IsMallocSet = true;
	}
}


__device__
void NCMCSS_PrintArray(float* array, int volume, int rows, int columns)
{
 
    for(int volumeIndex = 0; volumeIndex < volume; volumeIndex++)
    {
        //#pragma unroll 5
        for(int rowIndex = 0; rowIndex < rows; rowIndex++)
        {
            //#pragma unroll 10
            for(int columnIndex = 0; columnIndex < columns; columnIndex++)
            {
                printf("%f ", ELEMENT(array, volumeIndex, rowIndex, columnIndex, rows, columns));
            }
            printf("\n");
        }  
         printf("*************\n"); 
    } 
}

__device__
void NCMCSS_setArrayContentsToZero(float* array, int numRows, int numColumns)
{
    for(int rowIndex = 0; rowIndex < numRows; rowIndex++)
    {
        for(int columnIndex = 0; columnIndex < numColumns; columnIndex++)
        {
            ELEMENT(array, 0, rowIndex, columnIndex, numRows, numColumns) = 0;
        }
    }

}

__device__ void NCMCSS_getPatchWithMidPoint(float* input, int layerNumber, int rowNumber, int columnNumber, int numLayers, int mapHeight, int mapWidth, int patchwidth, float* focusPatch)
{
    int halfPatchWidth = patchwidth / 2;

    #pragma unroll 5
    for(int rowIndex = -halfPatchWidth; rowIndex <= halfPatchWidth; rowIndex++)
    {
        #pragma unroll 5
        for(int columnIndex = -halfPatchWidth; columnIndex <= halfPatchWidth; columnIndex++)
        {
            int currentMapRowIndex = rowIndex + rowNumber;
            int currentMapColumnIndex = columnIndex + columnNumber;
            
            int focusPatchRowIndex = rowIndex + halfPatchWidth;
            int focusPatchColumnIndex = columnIndex + halfPatchWidth;
                      
            ELEMENT(focusPatch, 0, focusPatchRowIndex, focusPatchColumnIndex, patchwidth, patchwidth) = 0;
            
            if(currentMapRowIndex >= 0 && currentMapRowIndex < mapHeight &&
                currentMapColumnIndex >= 0 && currentMapColumnIndex < mapWidth)
            {
                ELEMENT(focusPatch, 0, focusPatchRowIndex, focusPatchColumnIndex, patchwidth, patchwidth) = 
                        ELEMENT(input, layerNumber, currentMapRowIndex, currentMapColumnIndex, mapHeight, mapWidth);
            }
        }
    }
}


__device__
void NCMCSS_getMeanAndStd(float* meanMaps, float* stdMaps, int mapHeight, int mapWidth, int patchwidth, int layerNumber, int rowNumber, int columnNumber, float* mean, float* std)
{
	*mean = ELEMENT(meanMaps, layerNumber, rowNumber + 2, columnNumber + 2, mapHeight + 4, mapWidth + 4);
	*std = ELEMENT(stdMaps, layerNumber, rowNumber + 2, columnNumber + 2, mapHeight + 4, mapWidth + 4);
}

__device__ float NCMCSS_correlatePatch(float* input, int layerNumber, int rowNumber, int columnNumber, int numLayers, int mapHeight, int mapWidth, int patchwidth, int verticalWidth, float* focusPatch, float correlatorMean, float correlatorStd, float* meanMaps, float* stdMaps)
{
    int halfPatchWidth = patchwidth / 2;
    float correlatedValue = 0;
    int N = (patchwidth * patchwidth);
    
    float correlateeMean = 0, correlateeStd = 0;
    
    //get the bottom layer patch's mean and std
    NCMCSS_getMeanAndStd(meanMaps, stdMaps, mapHeight, mapWidth, patchwidth, layerNumber, rowNumber, columnNumber, &correlateeMean, &correlateeStd);

    //calculate normalization constant
    float normConstant = 1 / ((N-1) * (correlateeStd * correlatorStd));
      
    #pragma unroll 5
    for(int rowIndex = -halfPatchWidth; rowIndex <= halfPatchWidth; rowIndex++)
    {
        #pragma unroll 5
        for(int columnIndex = -halfPatchWidth; columnIndex <= halfPatchWidth; columnIndex++)
        {
			float correlatorVal = 0, correlateeVal = 0;
			
            int currentMapRowIndex = rowIndex + rowNumber;
            int currentMapColumnIndex = columnIndex + columnNumber;
            
            int focusPatchRowIndex = rowIndex + halfPatchWidth;
            int focusPatchColumnIndex = columnIndex + halfPatchWidth;
            
            correlatorVal = ELEMENT(focusPatch, 0, focusPatchRowIndex, focusPatchColumnIndex, patchwidth, patchwidth);
            
            if(currentMapRowIndex >= 0 && currentMapRowIndex < mapHeight &&
                currentMapColumnIndex >= 0 && currentMapColumnIndex < mapWidth)
            {
                 correlateeVal = ELEMENT(input, layerNumber, currentMapRowIndex, currentMapColumnIndex, mapHeight, mapWidth);
            }
            
			correlatedValue += (((correlatorVal - correlatorMean) * (correlateeVal - correlateeMean)) * normConstant);
        }
    }
    
    return correlatedValue;
}

__device__ 
void NCMCSS_ConstrainedCorrelation(float* input, float* focusPatch, int layerNumber, int rowNumber, int columnNumber, int numLayers, int mapHeight, int mapWidth, float* output, int patchwidth, int verticalWidth, float* meanMaps, float* stdMaps)
{
    //here outputRowNumber is dummy parameter now. since 37x12x5 =~ 2000 threads are not allowed to spawn in CUDA
    int halfVerticalWidth = verticalWidth / 2;
    int halflayerspoint = numLayers / 2;
    float correlatorMean = 0, correlatorStd = 0;
    
    //get the correlator mean and std
    NCMCSS_getMeanAndStd(meanMaps, stdMaps, mapHeight, mapWidth, patchwidth, layerNumber - halflayerspoint, rowNumber, columnNumber, &correlatorMean, &correlatorStd);

    for(int rowIndex = 0; rowIndex < verticalWidth; rowIndex++)
    { 
        int currentRowIndex = rowNumber - halfVerticalWidth + rowIndex;
                
        //in this particular row, for each column element
        //#pragma unroll 12
        for(int columnIndex = 0; columnIndex < verticalWidth; columnIndex++)
        {
            int currentColumnIndex = columnNumber - halfVerticalWidth + columnIndex;
            
            float correlatedValue = NCMCSS_correlatePatch(input, layerNumber, currentRowIndex, currentColumnIndex, numLayers, mapHeight, mapWidth, patchwidth, verticalWidth, focusPatch, correlatorMean, correlatorStd, meanMaps, stdMaps);

            //store the correlatedvalue in output
            ELEMENT(output, (layerNumber - halflayerspoint) * verticalWidth * verticalWidth + rowIndex * verticalWidth + columnIndex, rowNumber, columnNumber, mapHeight, mapWidth) = correlatedValue;

        }
    }

}


__global__
void NCMCSS_calcNormCrossMapCorrelation(float* input, int numLayers, int mapHeight, int mapWidth, float* output, int patchwidth, int verticalWidth, float* meanMaps, float* stdMaps)
{
    //calculate the layer index, row index, column index    
    int blockNumber = blockIdx.x;
    int elementIndex = threadIdx.x;
    
    //determine source input row number , column number
    int layerNumber = blockNumber;
    int inputRowNumber = elementIndex / mapWidth;
    int inputColumnNumber = elementIndex % mapWidth;

   // if(!(layerNumber == 0 && rowNumber == 0 && columnNumber == 0)) return;
    int halflayerspoint = numLayers / 2;

    // get the particular focus patch
    float focusPatch[25];
    NCMCSS_setArrayContentsToZero(focusPatch, patchwidth, patchwidth);
    
    NCMCSS_getPatchWithMidPoint(input, layerNumber, inputRowNumber, inputColumnNumber, numLayers, mapHeight, mapWidth, patchwidth, focusPatch);

    // for each element of its neighborhood rows, calculate the patch correlation
    // for this particular element, calculate the constrained crosspatch correlation
    // and save it in 'output' buffer
    //float* input, float* focusPatch, int layerNumber, int rowNumber, int columnNumber, int numLayers, int mapHeight, int mapWidth, float* output, int patchwidth, int verticalWidth
    NCMCSS_ConstrainedCorrelation(input, focusPatch, layerNumber + halflayerspoint, inputRowNumber, inputColumnNumber, numLayers, mapHeight, mapWidth, output, patchwidth, verticalWidth, meanMaps, stdMaps);

}

__global__
void NCMCSS_calcMeanAndStdMaps(float* input, int numLayers, int mapHeight, int mapWidth, int patchwidth, int verticalWidth, float* meanMaps, float* stdMaps) 
{
    //calculate the layer index, row index, column index    
    int blockNumber = blockIdx.x;
    int elementIndex = threadIdx.x;
    
    //determine source input row number , column number
    int layerNumber = blockNumber;
    int inputRowNumber = elementIndex / (mapWidth + 4);
    int inputColumnNumber = elementIndex % (mapWidth + 4);
    int halfVericalWidth = verticalWidth / 2;

    float focusPatch[25];
    NCMCSS_setArrayContentsToZero(focusPatch, patchwidth, patchwidth);
    
    NCMCSS_getPatchWithMidPoint(input, layerNumber, inputRowNumber - halfVericalWidth, inputColumnNumber - halfVericalWidth, numLayers, mapHeight, mapWidth, patchwidth, focusPatch);
    
    float sum = 0;
    
    // calculate the mean and standard deviation
    for(int index = 0; index < patchwidth * patchwidth; index++)
    {
		sum += focusPatch[index];
	}
	
	// calculate the mean
	//mapHeight + 4 , since the 2 rows above and 2 rows below are included in correlation
    ELEMENT(meanMaps, layerNumber, inputRowNumber, inputColumnNumber, mapHeight + 4, mapWidth + 4) = sum / (patchwidth * patchwidth);
	float mean = ELEMENT(meanMaps, layerNumber, inputRowNumber, inputColumnNumber, mapHeight + 4, mapWidth + 4);
	float variance = 0;
	
    // calculate the mean and standard deviation
    for(int index = 0; index < patchwidth * patchwidth; index++)
    {
		variance += ((focusPatch[index] - mean) * (focusPatch[index] - mean));
	}
	
    //mapHeight + 4 , since the 2 rows above and 2 rows below are included in correlation + eps value previous: 1e-6
    float std = sqrt(variance / ((patchwidth * patchwidth) - 1)) + 1e-2;
	ELEMENT(stdMaps, layerNumber, inputRowNumber, inputColumnNumber, mapHeight + 4, mapWidth + 4) = std;
}

void NCMCSS_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *output, int patchwidth, int verticalWidth, THCudaTensor *meanMaps, THCudaTensor *stdMaps) {

	THCUNN_assertSameGPU(state, 2, input, output);

	THArgCheck(input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch mode) tensor is expected");

	int numLayers = input->size[0];
	int mapHeight = input->size[1];
	int mapWidth = input->size[2];
	
	//get c-pointer address
	float* inputContents = THCudaTensor_data(state, input);

    //set malloc limit to a higher limit
	//setMallocLimit();

	//resize output
    int outputNumLayers = (numLayers / 2) * verticalWidth * verticalWidth;
    int outputMapHeight =  mapHeight;
    int outputMapWidth = mapWidth;

	// assign memory for mean, std maps
	THCudaTensor_resize3d(state, meanMaps, numLayers, mapHeight + 4, mapWidth + 4);
	THCudaTensor_fill(state, meanMaps, 0);
	THCudaTensor_resize3d(state, stdMaps, numLayers, mapHeight + 4, mapWidth + 4);
	THCudaTensor_fill(state, stdMaps, 0);
	
	//get c-pointer address
	float* meanMapsPtr = THCudaTensor_data(state, meanMaps);
	float* stdMapsPtr = THCudaTensor_data(state, stdMaps);
	
	int totalBlocks = numLayers;
	int totalThreads = (mapHeight + 4) * (mapWidth + 4);
	
	NCMCSS_calcMeanAndStdMaps<<<totalBlocks, totalThreads>>>(inputContents, numLayers, mapHeight, mapWidth, patchwidth, verticalWidth, meanMapsPtr, stdMapsPtr);
		
	THCudaTensor_resize3d(state, output, outputNumLayers, outputMapHeight, outputMapWidth);
    THCudaTensor_fill(state, output, 0);

	//get c-pointer address
	float* outputPtr = THCudaTensor_data(state, output);

	totalBlocks = (numLayers / 2);
	totalThreads = mapHeight * mapWidth;

    NCMCSS_calcNormCrossMapCorrelation<<<totalBlocks, totalThreads>>>(inputContents, numLayers, mapHeight, mapWidth, outputPtr, patchwidth, verticalWidth, meanMapsPtr, stdMapsPtr);

}

__device__
float NCMCSS_getCorrelationValue(float* output, int layerIndex, int mapHeight, int mapWidth, int verticalWidth, int relativeCorrelateeRowIndex, int relativeCorrelateeColumnIndex, int currentMapRowIndex, int currentMapColumnIndex)
{
	float correlationValue = 0;
	correlationValue = ELEMENT(output, (layerIndex * verticalWidth * verticalWidth) + (relativeCorrelateeRowIndex * verticalWidth) + relativeCorrelateeColumnIndex, 
                                currentMapRowIndex, currentMapColumnIndex, mapHeight, mapWidth);
    
	return correlationValue;
}

__device__
void NCMCSS_retrieveOutputGradient(float* gradOutput, float* currentOutputGradient, int layerNumber, int currentMapRowIndex, int currentMapColumnIndex, int mapHeight, int mapWidth, int patchwidth, int verticalWidth)
{
	int startLayerNumber = layerNumber * verticalWidth * verticalWidth;
    //#pragma unroll 5
    for(int rowIndex = 0; rowIndex < verticalWidth; rowIndex++)
    {
        //#pragma unroll 12
        for(int columnIndex = 0; columnIndex < verticalWidth; columnIndex++)
        {
        	int currentLayerNumber = startLayerNumber + (rowIndex * verticalWidth) + columnIndex;
            ELEMENT(currentOutputGradient, 0, rowIndex, columnIndex, verticalWidth, verticalWidth) = ELEMENT(gradOutput, currentLayerNumber, currentMapRowIndex, currentMapColumnIndex, mapHeight, mapWidth);
        }
        
    }
}

__device__
float NCMCSS_retrieveAndMultiplyOutputGradWithCorrelator(float* gradOutput, float* currentCorrelator, int layerNumber, int currentMapRowIndex, int currentMapColumnIndex, int mapHeight, int mapWidth, int patchwidth, int verticalWidth)
{
	int startLayerNumber = layerNumber * verticalWidth * verticalWidth;
	float inputGradient = 0;

    #pragma unroll 5
    for(int rowIndex = 0; rowIndex < verticalWidth; rowIndex++)
    {
        #pragma unroll 12
        for(int columnIndex = 0; columnIndex < verticalWidth; columnIndex++)
        {
        	int currentLayerNumber = startLayerNumber + (rowIndex * verticalWidth) + columnIndex;
            inputGradient += ELEMENT(currentCorrelator, 0, rowIndex, columnIndex, verticalWidth, verticalWidth) *
            		ELEMENT(gradOutput, currentLayerNumber, currentMapRowIndex, currentMapColumnIndex, mapHeight, mapWidth);
        }

    }

    return inputGradient;
}

__device__
void NCMCSS_gradientWrtnput(float* input, float* output, float* currentGradInput, int currentMapRowIndex, int currentMapColumnIndex, int layerNumber, int wrtRowNumber, int wrtColumnNumber, int numLayers, int mapHeight, int mapWidth, int patchwidth, int verticalWidth, float* meanMaps, float* stdMaps)
{
    int halfPatchWidth = patchwidth / 2;
    int halfVerticalWidth = verticalWidth / 2;
    int halflayerspoint = numLayers / 2;
    
    // currentMapColumnIndex = column of the current focus patch's mid pixel
    // wrtColumnNumber = the static pixel point whose gradient we are finding out
    // halfPatchWidth = half of the patch width
    //  0 0 0 0 0
    //  0 0 0 0 0
    //  0 0 x 0 0 
    //  0 0 0 0 0
    //  0 0 0 0 0
    // example : 1 - 3 + 2 = 0
    //           5 - 3 + 2 = 4    
        
    // gradient w.r.t., x_i = (((y_i - y_bar) / sigma_y) - (corr(x,y) * (x_i - x_bar) / sigma_x)) / ((N-1) * sigma_x)
    float N = patchwidth * patchwidth;
    float x_i = ELEMENT(input, layerNumber - halflayerspoint, wrtRowNumber, wrtColumnNumber, mapHeight, mapWidth);
    float sigma_x = 0, x_bar = 0;
    NCMCSS_getMeanAndStd(meanMaps, stdMaps, mapHeight, mapWidth, patchwidth, layerNumber - halflayerspoint, currentMapRowIndex, currentMapColumnIndex, &x_bar, &sigma_x);
    float gradX = (x_i - x_bar) / sigma_x;
    float outsideConstant = 1 / ((N-1) * sigma_x);
    int corrTorTeeRowDiff = currentMapRowIndex - wrtRowNumber;
    int corrTorTeeColumnDiff = currentMapColumnIndex - wrtColumnNumber;
    
    //#pragma unroll 5
    //here the range of values for gradient calculation is rowNumber+-2, columnNumber+-2 (5x5 matrix)
    for(int rowIndex = 0; rowIndex < verticalWidth; rowIndex++)
    {
        //#pragma unroll 12
        for(int columnIndex = 0; columnIndex < verticalWidth; columnIndex++)
        {
            int currentRowIndex = wrtRowNumber - halfVerticalWidth + rowIndex;
            int currentColumnIndex = wrtColumnNumber - halfVerticalWidth + columnIndex;
            float y_i = 0, sigma_y = 0, y_bar = 0;
            
            if(currentRowIndex >= 0 && currentRowIndex < mapHeight && currentColumnIndex >= 0 && currentColumnIndex < mapWidth) 
            {
				// here layerNumber > halflayerspoint (i.e., bottom layer)
            	y_i = ELEMENT(input, layerNumber, currentRowIndex, currentColumnIndex, mapHeight, mapWidth);
            }
            
            NCMCSS_getMeanAndStd(meanMaps, stdMaps, mapHeight, mapWidth, patchwidth, layerNumber, currentRowIndex  + corrTorTeeRowDiff,
                                                                                                 currentColumnIndex + corrTorTeeColumnDiff, &y_bar, &sigma_y);
            
            float corrValue = NCMCSS_getCorrelationValue(output, layerNumber - halflayerspoint, mapHeight, mapWidth, verticalWidth, 
												rowIndex, columnIndex, 
												currentMapRowIndex, currentMapColumnIndex);
            //currentRowIndex + corrTorTeeRowDiff gives the row w.r.t. which correlation is carried out
            //currentRowIndex + corrTorTeeRowDiff - currentMapRowIndex  gives the difference between the wrt-correlation-row and actual midpoint of patch
            //currentRowIndex + corrTorTeeRowDiff - currentMapRowIndex + halfPatchWidth gives any of 0,1,2,3,4
            
            ELEMENT(currentGradInput, 0, rowIndex, columnIndex, verticalWidth, verticalWidth) = (((y_i - y_bar) / sigma_y) - (corrValue * gradX)) * outsideConstant;
        }        
    }
    
}

__device__
void NCMCSS_getGradientForTopLayer(float* input, float* output, int layerNumber, int rowNumber, int columnNumber, int numLayers, int mapHeight, int mapWidth, float* gradOutput, float* gradInput, int patchwidth, int verticalWidth, float* meanMaps, float* stdMaps)
{
    int halfPatchWidth = patchwidth / 2;
    int halflayerspoint = numLayers / 2;
    float inputGradient = 0;
    
    //float* currentCorrelator = new float[verticalWidth * mapWidth];
    float currentCorrelator[25];

    //column index is placed first, as it gives advantage of retrieving output-gradient only once per column
    #pragma unroll 9
    for(int columnIndex = -halfPatchWidth; columnIndex <= halfPatchWidth; columnIndex++)
    {
        // calculate current column number
        int currentMapColumnIndex = columnNumber + columnIndex;

    	#pragma unroll 9
    	for(int rowIndex = -halfPatchWidth; rowIndex <= halfPatchWidth; rowIndex++)
        {
            int currentMapRowIndex = rowNumber + rowIndex;
            
           //if the row and column index is valid, get the corresponding gradient
            if(currentMapRowIndex >= 0 && currentMapRowIndex < mapHeight and
                currentMapColumnIndex >= 0 and currentMapColumnIndex < mapWidth)
            {
                NCMCSS_setArrayContentsToZero(currentCorrelator, verticalWidth, verticalWidth);
                NCMCSS_gradientWrtnput(input, output, currentCorrelator, currentMapRowIndex, currentMapColumnIndex, 
                                    layerNumber + halflayerspoint, rowNumber, columnNumber, numLayers, mapHeight, mapWidth,
                                    patchwidth, verticalWidth, meanMaps, stdMaps);
                
                //retrieve the output gradient for currentMapRowIndex, currentMapColumnIndex
            	inputGradient += NCMCSS_retrieveAndMultiplyOutputGradWithCorrelator(gradOutput, currentCorrelator, layerNumber, currentMapRowIndex, currentMapColumnIndex, mapHeight, mapWidth, patchwidth, verticalWidth);
                
            }
        }
    }
    
    ELEMENT(gradInput, layerNumber, rowNumber, columnNumber, mapHeight, mapWidth) = inputGradient;

}


__device__
float NCMCSS_retrieveGradientWrtCorrelateeInput(float* input, float* output, float* currentGradientForBottom, int layerNumber, int currentRowIndex, 
												int currentColumnIndex, int rawRowIndex, int rawColumnIndex, int rowNumber, int columnNumber, int numLayers, int mapHeight,
												int mapWidth, int patchwidth, int verticalWidth, float* meanMaps, float* stdMaps)
{
	float focusPatch[25];
    NCMCSS_setArrayContentsToZero(focusPatch, patchwidth, patchwidth);
    int halfPatchWidth = patchwidth / 2;
    int halflayerspoint = numLayers / 2;
    
    NCMCSS_getPatchWithMidPoint(input, layerNumber, currentRowIndex, currentColumnIndex, numLayers, mapHeight, mapWidth, patchwidth, focusPatch);
    float x_bar = 0, sigma_x = 0;
    //currentRowIndex, currentColumnIndex = the top layer focus patch center
    NCMCSS_getMeanAndStd(meanMaps, stdMaps, mapHeight, mapWidth, patchwidth, layerNumber, currentRowIndex, currentColumnIndex, &x_bar, &sigma_x);
    
    //determine start index
    int startIndex = 0;
    if(rawRowIndex < 0)
    {
        startIndex = -rawRowIndex;
    }
    else 
    {
        startIndex = 0;
    }
                
    //determine end index
    int endIndex = 0;
    if(rawRowIndex < 0)
    {
        endIndex = patchwidth - 1;
    }
    else{
        endIndex = patchwidth - rawRowIndex - 1;
    }
     
    float inputGradient = 0;
    float N = (patchwidth * patchwidth);
            
    //#pragma unroll 5
    for(int reverseRowIndex = startIndex; reverseRowIndex <= endIndex; reverseRowIndex++)
    {
        int focusPatchRowIndex = endIndex - (reverseRowIndex - startIndex);
        
        for(int reverseColumnIndex = 0; reverseColumnIndex < patchwidth; reverseColumnIndex++)
        {
            int focusPatchColumnIndex = patchwidth - reverseColumnIndex - 1;
            int inputGradientColumnNumber = (columnNumber + reverseColumnIndex - halfPatchWidth);
            int bottomCentreRowIndex = rowNumber - (focusPatchRowIndex - halfPatchWidth);
            
            if(inputGradientColumnNumber >= currentColumnIndex - halfPatchWidth && inputGradientColumnNumber <= currentColumnIndex + halfPatchWidth
                && bottomCentreRowIndex >= currentRowIndex - halfPatchWidth && bottomCentreRowIndex <= currentRowIndex + halfPatchWidth) {
				// get the bottom layer's patchmidpoint
				// the rowNumber is the row at which the influential grad elements of focusPatch is there right now.
				// focusPatchRowIndex contains one of (0 1 2 3 4)
				// simply, if focusPatchRowIndex = 2, then bottomCentreRowIndex = rowNumber. go by this intuition				
                
                float corrValue = NCMCSS_getCorrelationValue(output, layerNumber, mapHeight, mapWidth, verticalWidth, 
                                                        // currentRowIndex = -4 to 4 row numbers surrounding bottom layer element
                                                        bottomCentreRowIndex - currentRowIndex + halfPatchWidth, inputGradientColumnNumber - currentColumnIndex + halfPatchWidth,
															currentRowIndex, currentColumnIndex);
                
                float x_i = ELEMENT(focusPatch, 0, focusPatchRowIndex, focusPatchColumnIndex, patchwidth, patchwidth);
                float gradX = (x_i - x_bar) / sigma_x;
                
                // determine y related values
                float y_i = ELEMENT(input, layerNumber + halflayerspoint, rowNumber, columnNumber, mapHeight, mapWidth);
                float y_bar = 0, sigma_y = 0;
                NCMCSS_getMeanAndStd(meanMaps, stdMaps, mapHeight, mapWidth, patchwidth, layerNumber + halflayerspoint, 
                                    bottomCentreRowIndex, inputGradientColumnNumber, &y_bar, &sigma_y);
                
                float outsideConstant = 1/((N-1) * sigma_y);
                float currentGradInput = (gradX - (corrValue * (y_i - y_bar) / sigma_y)) * outsideConstant;
				inputGradient += currentGradInput * ELEMENT(currentGradientForBottom, 0, reverseRowIndex, inputGradientColumnNumber - currentColumnIndex + halfPatchWidth, verticalWidth, verticalWidth);
            }
        }
    }    
    
    //delete[] focusPatch;
    return inputGradient;
}

__device__
void NCMCSS_getGradientForBottomLayer(float* input, float* output, int layerNumber, int rowNumber, int columnNumber, int numLayers, int mapHeight, int mapWidth, float* gradOutput, float* gradInput, int patchwidth, int verticalWidth, float* meanMaps, float* stdMaps)
{
    int halfPatchWidth = patchwidth / 2;
    int halfVerticalWidth = verticalWidth / 2;
    int halflayerspoint = numLayers / 2;
    float inputGradient = 0;
    //float* currentOutputGradient = new float[verticalWidth * mapWidth];
    float currentOutputGradient[5 * 5];
  
    //#pragma unroll 9
    for(int rowIndex = -halfVerticalWidth * 2; rowIndex <= halfVerticalWidth * 2; rowIndex++)
    {
        //#pragma unroll 12
        for(int columnIndex = -halfVerticalWidth * 2; columnIndex <= halfVerticalWidth * 2; columnIndex++)
        {
            int currentRowIndex = rowNumber + rowIndex;
            int currentColumnIndex = columnNumber + columnIndex; 
            
           //if the row and column index is valid, get the corresponding gradient
            if(currentRowIndex >= 0 && currentRowIndex < mapHeight &&
                currentColumnIndex >= 0 && currentColumnIndex < mapWidth) 
            {
                NCMCSS_setArrayContentsToZero(currentOutputGradient, verticalWidth, verticalWidth);
                
                //retrieve the output gradient for currentMapRowIndex, currentMapColumnIndex
                NCMCSS_retrieveOutputGradient(gradOutput, currentOutputGradient, layerNumber, currentRowIndex, currentColumnIndex, mapHeight, mapWidth, patchwidth, verticalWidth);
                
                // retrieve the gradient with respect to the input for current respective element
                //currentRowIndex, currentColumnIndex - to get the focus patch with corresponding mid point
                inputGradient += NCMCSS_retrieveGradientWrtCorrelateeInput(input, output, currentOutputGradient, layerNumber, currentRowIndex, 
																		currentColumnIndex, rowIndex, columnIndex, rowNumber, columnNumber, numLayers, mapHeight, 
																		mapWidth, patchwidth, verticalWidth, meanMaps, stdMaps);
            }
        }
    }
    
    //delete[] currentOutputGradient;

    ELEMENT(gradInput, layerNumber + halflayerspoint, rowNumber, columnNumber, mapHeight, mapWidth) = inputGradient;
}

__global__
void NCMCSS_calcNormCrossMapCorrelationGradInput(float* input, float* output, int numLayers, int mapHeight, int mapWidth, float* gradOutput, float* gradInput, int patchwidth, int verticalWidth, float* meanMaps, float* stdMaps)
{
    //calculate the layer index, row index, column index
    int layerNumber = blockIdx.x;
    int elementIndex = threadIdx.x;
    int halflayerspoint = numLayers / 2;
    
    int rowNumber = elementIndex / mapWidth;
    int columnNumber = elementIndex % mapWidth;
    
    //if(!(layerNumber == 2 && rowNumber == 3 && columnNumber == 0)) return;
    
    if(layerNumber < halflayerspoint) {
        NCMCSS_getGradientForTopLayer(input, output, layerNumber, rowNumber, columnNumber, numLayers, mapHeight, mapWidth, gradOutput, gradInput, patchwidth, verticalWidth, meanMaps, stdMaps);
    }
    else
    {
        layerNumber = layerNumber - halflayerspoint;
        NCMCSS_getGradientForBottomLayer(input, output, layerNumber, rowNumber, columnNumber, numLayers, mapHeight, mapWidth, gradOutput, gradInput, patchwidth, verticalWidth, meanMaps, stdMaps);
    }

}

void NCMCSS_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *output, THCudaTensor *gradOutput, THCudaTensor *gradInput, int patchwidth, int verticalWidth, THCudaTensor *meanMaps, THCudaTensor *stdMaps) {

	//determine gradInput sizes
	int numLayers = input->size[0];
	int mapHeight = input->size[1];
	int mapWidth = input->size[2];

	//resize gradInputs
	THCudaTensor_resize3d(state, gradInput, numLayers, mapHeight, mapWidth);
    THCudaTensor_fill(state, gradInput, 0);

	//get elementary datatype pointers
	float* inputContents = THCudaTensor_data(state, input);
	float* outputContents = THCudaTensor_data(state, output);
	float* gradOutputPtr = THCudaTensor_data(state, gradOutput);
	float* gradInputPtr = THCudaTensor_data(state, gradInput);
	float* meanMapsPtr = THCudaTensor_data(state, meanMaps);
	float* stdMapsPtr = THCudaTensor_data(state, stdMaps);

    NCMCSS_calcNormCrossMapCorrelationGradInput<<<numLayers, mapHeight * mapWidth>>>(inputContents, outputContents, numLayers, mapHeight, mapWidth, gradOutputPtr, gradInputPtr, patchwidth, verticalWidth, meanMapsPtr, stdMapsPtr);
    CudaCheckError();
}

}
